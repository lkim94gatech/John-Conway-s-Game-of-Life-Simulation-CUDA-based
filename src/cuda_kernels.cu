#include "hip/hip_runtime.h"
/**
 * Leandro Alan Kim
 * Class: ECE 4122 A
 * Last Modified: 8 Nov 2024
 * 
 * Implement a C++ CUDA program to run the Game of Life.
 */

#include <SFML/Graphics.hpp>
#include <iostream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace sf;
using namespace std;

// default window
int WINDOW_WIDTH = 800; // default width
int WINDOW_HEIGHT = 600; // default height
int PIXEL_SIZE = 5; // default cell size
int threadsPerBlock = 32; // default threads/block
string MEMORY_TYPE = "NORMAL"; // default memory type

// calculating grid
inline int GRID_WIDTH()
{
    return WINDOW_WIDTH / PIXEL_SIZE;
}
inline int GRID_HEIGHT()
{
    return WINDOW_HEIGHT / PIXEL_SIZE;
}

/**
 * count neighbors that are alive
 * @param grid current grid
 * @param x x coordinate
 * @param y y coordinate
 * @param width grid width
 * @param height grid height
 * @return number of cells that are alive neighboring
 */
__device__ int countNeighbors(unsigned char* grid, int x, int y, int width, int height)
{
    int count = 0;
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            if (i == 0 && j == 0) continue; // not counting itself
            // neighbor coordinates
            int nx = (x + i + width) % width;
            int ny = (y + j + height) % height;
            count += grid[ny * width + nx];
        }
    }
    return count;
}

/**
 * update grid
 * @param currentGrid current grid
 * @param nextGrid next grid
 * @param width grid width
 * @param height grid height
 */
__global__ void updateGridKernel(unsigned char* currentGrid, unsigned char* nextGrid, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    int neighbors = countNeighbors(currentGrid, x, y, width, height);
    
    if (currentGrid[idx]) {
        nextGrid[idx] = (neighbors == 2 || neighbors == 3) ? 1 : 0;
    } else {
        nextGrid[idx] = (neighbors == 3) ? 1 : 0;
    }
}

/**
 * random number generator to set alive and dead
 * @param grid grid to start
 * @param width grid width
 * @param height grid height
 * @param seed random
 */
__global__ void initGridKernel(unsigned char* grid, int width, int height, unsigned long seed)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    hiprandState state;
    hiprand_init(seed + y * width + x, 0, 0, &state);
    grid[y * width + x] = (hiprand_uniform(&state) < 0.5f) ? 1 : 0; // 50% to be alive
}

/**
 * normal memory- pageable
 * @param h_grid host grid pointer
 * @param d_currentGrid device current grid pointer
 * @param d_nextGrid device next grid pointer
 * @param byteSize size of memory
 */
void allocateNormalMemory(unsigned char*& h_grid, unsigned char*& d_currentGrid, unsigned char*& d_nextGrid, size_t byteSize)
{
    h_grid = new unsigned char[byteSize];
    memset(h_grid, 0, byteSize);
    hipMalloc(&d_currentGrid, byteSize);
    hipMalloc(&d_nextGrid, byteSize);
    hipMemset(d_currentGrid, 0, byteSize);
    hipMemset(d_nextGrid, 0, byteSize);
}

/**
 * pinned memory
 * @param h_grid host grid pointer
 * @param d_currentGrid device current grid pointer
 * @param d_nextGrid device next grid pointer
 * @param byteSize size of memory
 */
void allocatePinnedMemory(unsigned char*& h_grid, unsigned char*& d_currentGrid, unsigned char*& d_nextGrid, size_t byteSize)
{
    hipHostMalloc(&h_grid, byteSize);  // Pinned memory allocation
    memset(h_grid, 0, byteSize);
    hipMalloc(&d_currentGrid, byteSize);
    hipMalloc(&d_nextGrid, byteSize);
    hipMemset(d_currentGrid, 0, byteSize);
    hipMemset(d_nextGrid, 0, byteSize);
}

/**
 * managed memory - unified memory
 * @param h_grid host grid pointer
 * @param d_currentGrid device current grid pointer
 * @param d_nextGrid device next grid pointer
 * @param byteSize size of memory
 */
void allocateManagedMemory(unsigned char*& h_grid, unsigned char*& d_currentGrid, unsigned char*& d_nextGrid, size_t byteSize)
{
    hipMallocManaged(&d_currentGrid, byteSize);
    hipMallocManaged(&d_nextGrid, byteSize);
    hipMemset(d_currentGrid, 0, byteSize);
    hipMemset(d_nextGrid, 0, byteSize);
    h_grid = d_currentGrid;  // pointer shared between cpu and gpu
}

/**
 * command line arg
 * @param argc number of arguments
 * @param argv array of argument strings
 */
void arg(int argc, char* argv[])
{
    for (int i = 1; i < argc; ++i)
    {
        string arg = argv[i];
        if (arg == "-n")
        {
            threadsPerBlock = stoi(argv[++i]);
            // if not multiple of 32 then default
            if (threadsPerBlock % 32 != 0)
            {
                threadsPerBlock = 32;
            }
        } else if (arg == "-c")
        {
            PIXEL_SIZE = stoi(argv[++i]);
            // cell size has to be min 1 or default
            if (PIXEL_SIZE < 1)
            {
                PIXEL_SIZE = 5;
            }
        } else if (arg == "-x")
        {
            WINDOW_WIDTH = stoi(argv[++i]);
        } else if (arg == "-y" && i + 1 < argc)
        {
            WINDOW_HEIGHT = stoi(argv[++i]);
        } else if (arg == "-t" && i + 1 < argc)
        {
            MEMORY_TYPE = argv[++i];
            if (MEMORY_TYPE != "NORMAL" && MEMORY_TYPE != "PINNED" && MEMORY_TYPE != "MANAGED")
            {
                MEMORY_TYPE = "NORMAL"; // default
            }
        }
    }
}

int main(int argc, char* argv[])
{
    arg(argc, argv);

    // window
    RenderWindow window(VideoMode(WINDOW_WIDTH, WINDOW_HEIGHT), "CUDA Game of Life");
    window.setFramerateLimit(60);

    // setup memory
    const int gridSize = GRID_WIDTH() * GRID_HEIGHT();
    unsigned char *d_currentGrid, *d_nextGrid, *h_grid;
    size_t byteSize = gridSize * sizeof(unsigned char);

    // allocate memory based on memory type taken in
    if (MEMORY_TYPE == "NORMAL")
    {
        allocateNormalMemory(h_grid, d_currentGrid, d_nextGrid, byteSize);
    } else if (MEMORY_TYPE == "PINNED")
    {
        allocatePinnedMemory(h_grid, d_currentGrid, d_nextGrid, byteSize);
    } else // managed
    {
        allocateManagedMemory(h_grid, d_currentGrid, d_nextGrid, byteSize);
    }

    // config cuda
    dim3 blockSize(threadsPerBlock, threadsPerBlock);
    dim3 numBlocks((GRID_WIDTH() + blockSize.x - 1) / blockSize.x, (GRID_HEIGHT() + blockSize.y - 1) / blockSize.y);

    // grid
    unsigned long seed = chrono::system_clock::now().time_since_epoch().count();
    initGridKernel<<<numBlocks, blockSize>>>(d_currentGrid, GRID_WIDTH(), GRID_HEIGHT(), seed);
    hipDeviceSynchronize();

    // copy for normal and pinned memory
    if (MEMORY_TYPE != "MANAGED")
    {
        hipMemcpy(h_grid, d_currentGrid, byteSize, hipMemcpyDeviceToHost);
    }

    // track
    using clock = chrono::high_resolution_clock;
    unsigned long numGenerations = 0;
    long long totalTime = 0;

    // main loop
    while (window.isOpen())
    {
        Event event;
        while (window.pollEvent(event))
        {
            if (event.type == Event::Closed || (event.type == Event::KeyPressed && event.key.code == Keyboard::Escape))
            {
                window.close();
            }
        }

        // start time for curr generation
        auto startTime = clock::now();

        // update grid for memory tyype
        if (MEMORY_TYPE == "MANAGED")
        {
            updateGridKernel<<<numBlocks, blockSize>>>(d_currentGrid, d_nextGrid, GRID_WIDTH(), GRID_HEIGHT());
            hipDeviceSynchronize();
        } else // for normal and pinned
        {
            updateGridKernel<<<numBlocks, blockSize>>>(d_currentGrid, d_nextGrid, GRID_WIDTH(), GRID_HEIGHT());
            hipDeviceSynchronize();
            hipMemcpy(h_grid, d_currentGrid, byteSize, hipMemcpyDeviceToHost);
        }

        // swap curr and next generation
        swap(d_currentGrid, d_nextGrid);

        // stop time
        auto endTime = clock::now();
        totalTime += chrono::duration_cast<chrono::microseconds>(endTime - startTime).count();
        numGenerations++;

        // print for every 100 generation and reset time
        if (numGenerations % 100 == 0)
        {
            cout << "100 generations took " << totalTime << " microsecs with "
                 << threadsPerBlock << " threads per block using " 
                 << MEMORY_TYPE << " memory allocation." << endl;
            totalTime = 0;
        }

        // render curr state
        window.clear(Color::Black);
        RectangleShape cell(Vector2f(PIXEL_SIZE - 1, PIXEL_SIZE - 1));
        cell.setFillColor(Color::White);

        // draw alive cells
        for (int y = 0; y < GRID_HEIGHT(); y++)
        {
            for (int x = 0; x < GRID_WIDTH(); x++)
            {
                if (h_grid[y * GRID_WIDTH() + x])
                {
                    cell.setPosition(x * PIXEL_SIZE, y * PIXEL_SIZE);
                    window.draw(cell);
                }
            }
        }

        window.display();
    }

    // cleanup
    if (MEMORY_TYPE == "MANAGED")
    {
        hipFree(d_currentGrid);
        hipFree(d_nextGrid);
    } else
    {
        hipFree(d_currentGrid);
        hipFree(d_nextGrid);
        if (MEMORY_TYPE == "PINNED")
        {
            hipHostFree(h_grid);
        } else
        {
            delete[] h_grid;
        }
    }

    return 0;
}